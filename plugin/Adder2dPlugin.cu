#include "hip/hip_runtime.h"
#include <cstring>
#include <vector>
#include "hip/hip_runtime.h"
#include "hip/hip_fp16.h"

#include "NvInfer.h"
#include "NvInferPlugin.h"

#include "PluginUtils.h"
#include "Adder2dPlugin.h"


//adder cuda kernel
template <typename Ftype, unsigned int blockSize>
__global__ void filterSum(int filterIdx,  int in_c, int in_h, int in_w, int filter_size, int n_filters; int stride, \
                          int padding, const Ftype* input, Ftype* output, const Ftype* weights)
{
    int out_h = (in_h + pad - size) / stride + 1;
    int out_w = (in_w + pad - size) / stride + 1;
    int out_c = n_filters;
    int out_idx = out_h * out_w * filterIdx + blockIdx.x;

    extern __shared__ int sdata[];

    int tid = threadIdx.x;
    int blockid = blockIdx.x;

    sdata[tid] = 0;

    int tid_x = (tid % (filter_size*filterSize)) % filter_size;
    int tid_y = (tid % (filter_size*filterSize)) / filter_size;
    int tid_z = tid / (filter_size*filterSize);

    int blockid_x = blockid % out_w;
    int blockid_y = blockid / out_h;


    Ftype input_val;
    if((tid_x - padding)<0 |  (tid_y - padding)<0){
        printf("tid:%d, block_idx:%d, tid_x:%d, tid_y:%d, tid_z:%d", tid, output_idx, tid_x, tid_y, tid_z);
        input_val = 0;
    }
    else{
        //selecting channel --> tid_z*(in_c*in_h)
        //selecting location using blockid and stride --> (blockid_y*stride) +
        int maped_input_idx = tid_z*(in_c*in_h) + (tid_y-padding)*in_w + (tid_x-padding);
        input_val = input[mapping_input_idx];
    }


    int n_weights = filterSize * filterSize * out_c;
    int weight_idx = n_weights*filterIdx + tid;

    if(tid < n_weights)
    {

        sdata[tid] += fabs(input_val - weights[weight_idx]);
    }
    __syncthreads();

    if (blockSize >= 512) { if (tid < 256) { sdata[tid] += sdata[tid + 256]; } __syncthreads(); }
    if (blockSize >= 256) { if (tid < 128) { sdata[tid] += sdata[tid + 128]; } __syncthreads(); }
    if (blockSize >= 128) { if (tid < 64) { sdata[tid] += sdata[tid + 64]; } __syncthreads(); }

    if (tid < 32)
    {
        if (blockSize >= 64) sdata[tid] += sdata[tid + 32];
        if (blockSize >= 32) sdata[tid] += sdata[tid + 16];
        if (blockSize >= 16) sdata[tid] += sdata[tid + 8];
        if (blockSize >= 8) sdata[tid] += sdata[tid + 4];
        if (blockSize >= 4) sdata[tid] += sdata[tid + 2];
        if (blockSize >= 2) sdata[tid] += sdata[tid + 1];
    }

    if (tid == 0) output[out_idx] = -sdata[0];
}

// for consistency I recommend all plugin have same namesapce and version
const char* G_PLUGIN_NAMESPACE = "_TRT";
const char* G_PLUGIN_VERSION = "1";
const char* G_ADDER2D_TYPE = "Adder2d";
const char* G_ADDER2D_NAME = "Adder2d_TRT"; //plugin_name = plugin_type + plugin_namespace


Adder2dPlugin::Adder2dPlugin(const nvinfer1::Weights *weights, int nbWeights, int filterSize, int nbFilters, int stride,
                             int padding) {
    mWeights = weights[0];
    mWeights.values = malloc(mWeights.count * type2size(mWeights.type));
    memcpy(const_cast<void *>(mWeights.values), weights[0].values, mWeights.count * type2size(mWeights.type));
    mNbWeights = nbWeights;
    mFilterSize = filterSize;
    mNbFilters = nbFilters;
    mStride = stride;
    mPadding = padding;

}

// create the plugin at runtime from a byte stream
Adder2dPlugin::Adder2dPlugin(const void *data, size_t length) {
    const char *d = static_cast<const char *>(data), *a = d;
    read<int>(d, mNbInputChannels);
    read<int>(d, mNbInputHeight);
    read<int>(d, mNbInputWidth);
    read<int>(d, mNbWeights);
    read<int>(d, mFilterSize);
    read<int>(d, mNbFilters);
    read<int>(d, mStride);
    read<int>(d, mPadding);
    read<nvinfer1::DataType>(d, mDataType);
    read<int64_t>(d, mWeights.count);
    read<nvinfer1::DataType>(d, mWeights.type);
    mWeights.values = nullptr;
    mWeights.values = malloc(mWeights.count * type2size(mWeights.type));
    memcpy(const_cast<void *>(mWeights.values), d, mWeights.count * type2size(mWeights.type));
    d = d + mWeights.count * type2size(mWeights.type);
    ASSERT(d == a + length);
}

size_t Adder2dPlugin::getSerializationSize() const {
    return sizeof(mNbInputChannels) + sizeof(mNbInputWidth) + sizeof(mNbInputHeight) + sizeof(mFilterSize) +
           sizeof(mNbFilters) + sizeof(mStride) + sizeof(mPadding) + sizeof(mDataType) + sizeof(mWeights.count) +
           sizeof(mWeights.type) + mWeights.count * type2size(mWeights.type);
}

void Adder2dPlugin::serialize(void *buffer) const {
    char *d = static_cast<char *>(buffer), *a = d;
    write(d, mNbInputChannels);
    write(d, mNbInputHeight);
    write(d, mNbInputWidth);
    write(d, mNbWeights);
    write(d, mFilterSize);
    write(d, mNbFilters);
    write(d, mStride);
    write(d, mPadding);
    write(d, mDataType);
    write(d, mWeights.count);
    write(d, mWeights.type);
    convertAndCopyToBuffer(d, mWeights, mWeights.type);
    ASSERT(d == a + getSerializationSize());
}

Adder2dPlugin::~Adder2dPlugin() {
    if (mWeights.values)
    {
        free(const_cast<void *>(mWeights.values));
        mWeights.values = nullptr;
    }

//    if (mDeviceKernel)
//    {
//        hipFree(mDeviceKernel);
//        mDeviceKernel = nullptr;
//    }
}

int Adder2dPlugin::getNbOutputs() const {
    return 1;
}

nvinfer1::Dims Adder2dPlugin::getOutputDimensions(int index, const nvinfer1::Dims* inputs, int nbInputDims) {
    if(index == 0) {
        // CHW
        nvinfer1::Dims dimsOutput;
        dimsOutput.nbDims = inputs->nbDims;
        std::cout << "Input nbDims:" << inputs->nbDims << std::endl;
        dimsOutput.d[0] = mNbFilters;
        dimsOutput.d[1] = (inputs->d[1] + 2 * mPadding - mFilterSize) / mStride + 1;
        dimsOutput.d[2] = (inputs->d[2] + 2 * mPadding - mFilterSize) / mStride + 1;

        std::cout << "InputDimention:" << inputs->d[0] << "," << inputs->d[1] << "," <<  inputs->d[2] << std::endl;
        std::cout << "getOutputDimensions:" << dimsOutput.d[0] << "," << dimsOutput.d[1] << "," <<  dimsOutput.d[2] << std::endl;
        return dimsOutput;
    } // else if(index == n) {
        // for other outputs if exists.
    // }
    else {
        ASSERT(false);
    }
}


bool Adder2dPlugin::supportsFormat(nvinfer1::DataType type, nvinfer1::PluginFormat format) const {
    return (type == nvinfer1::DataType::kFLOAT || type == nvinfer1::DataType::kHALF)
            && format == nvinfer1::PluginFormat::kNCHW;
}

void Adder2dPlugin::configureWithFormat(const nvinfer1::Dims* inputDims, int nbInputs,
                                        const nvinfer1::Dims* outputDims, int nbOutputs,
                                        nvinfer1::DataType type, nvinfer1::PluginFormat format,
                                        int maxBatchSize) {
    ASSERT((type == nvinfer1::DataType::kFLOAT || type == nvinfer1::DataType::kHALF)
            && format == nvinfer1::PluginFormat::kNCHW);
    mNbInputChannels = inputDims[0].d[0];
    mNbInputHeight = inputDims[0].d[1];
    mNbInputWidth = inputDims[0].d[2];
    mDataType = type;
}

int Adder2dPlugin::initialize() {
//    convertAndCopyToDeivce(mDeviceKernel, mWeights, mDataType);
    return 0;
}

void Adder2dPlugin::terminate() {
    if (mWeights.values)
    {
        free(const_cast<void *>(mWeights.values));
        mWeights.values = nullptr;
    }
//    if (mDeviceKernel)
//    {
//        hipFree(mDeviceKernel);
//        mDeviceKernel = nullptr;
//    }
}

size_t Adder2dPlugin::getWorkspaceSize(int maxBatchSize) const{
    return 0;
}

int Adder2dPlugin::enqueue(int batchSize, const void *const *inputs, void **outputs, void *workspace, hipStream_t stream)
{
//    const int count = batchSize * mNbInputChannels * mNbInputWidth * mNbInputHeight;
//    const int channels = mNbInputChannels;
//    const int dim = mNbInputWidth * mNbInputHeight;
//    const int div_factor = 1;
//    if (mDataType == nvinfer1::DataType::kFLOAT)
//    {
//        const float zerof{0.0f};
//        CUDA_CHECK(Forward_gpu(count, channels, dim,
//                            reinterpret_cast<const float *>(mDeviceKernel),
//                            reinterpret_cast<const float *>(inputs[0]),
//                            reinterpret_cast<float *>(outputs[0]),
//                            zerof,
//                            div_factor,
//                            stream));
//    }
//#ifdef FP16_PRELU
//    else
//    {
//        const __half zeroh = __half(0.0f);
//        CUDA_CHECK(Forward_gpu(count, channels, dim,
//                            reinterpret_cast<const __half *>(mDeviceKernel),
//                            reinterpret_cast<const __half *>(inputs[0]),
//                            reinterpret_cast<__half *>(outputs[0]),
//                            zeroh,
//                            div_factor,
//                            stream));
//    }
//#else
//    else
//    {
//        spdlog::error("fp16 prelu is unsupported");
//        ASSERT(false);
//    }
//#endif
    return 0;
}

const char *Adder2dPlugin::getPluginType() const {
    return G_ADDER2D_TYPE;
}

const char *Adder2dPlugin::getPluginVersion() const {
    return G_PLUGIN_VERSION;
}

void Adder2dPlugin::destroy() {
    delete this;
}

nvinfer1::IPluginV2* Adder2dPlugin::clone() const {
    return new Adder2dPlugin(&mWeights, mNbWeights, mFilterSize, mNbFilters, mStride, mPadding);
}

void Adder2dPlugin::setPluginNamespace(const char* pluginNamespace) {

}

const char* Adder2dPlugin::getPluginNamespace() const {
    return G_PLUGIN_NAMESPACE;
}




Adder2dPluginCreator::Adder2dPluginCreator()  {
    mPluginAttributes.emplace_back(nvinfer1::PluginField("weights", nullptr, nvinfer1::PluginFieldType::kFLOAT32, 1));
    mPluginAttributes.emplace_back(nvinfer1::PluginField("nbWeight", nullptr, nvinfer1::PluginFieldType::kINT32, 1));
    mPluginAttributes.emplace_back(nvinfer1::PluginField("filterSize", nullptr, nvinfer1::PluginFieldType::kINT32, 1));
    mPluginAttributes.emplace_back(nvinfer1::PluginField("nbFilters", nullptr, nvinfer1::PluginFieldType::kINT32, 1));
    mPluginAttributes.emplace_back(nvinfer1::PluginField("stride", nullptr, nvinfer1::PluginFieldType::kINT32, 1));
    mPluginAttributes.emplace_back(nvinfer1::PluginField("padding", nullptr, nvinfer1::PluginFieldType::kINT32, 1));
    mFC.nbFields = mPluginAttributes.size();
    mFC.fields = mPluginAttributes.data();
}

// return ADDER2D_PLUGIN_TYPE + ADDER2D_PLUGIN_NAMESPACE
const char* Adder2dPluginCreator::getPluginName() const {
    // std::string plugin_type{G_ADDER2D_TYPE};
    // std::string plugin_namespace{G_PLUGIN_NAMESPACE};
    // return (plugin_type+plugin_namespace).c_str();
    return G_ADDER2D_NAME;
}

const char* Adder2dPluginCreator::getPluginVersion() const {
    return G_PLUGIN_VERSION;
}

const nvinfer1::PluginFieldCollection* Adder2dPluginCreator::getFieldNames() {
    return &mFC;
}

nvinfer1::IPluginV2* Adder2dPluginCreator::createPlugin(const char* name, const nvinfer1::PluginFieldCollection* fc) {
    int nbWeights, filterSize, nbFilters, stride, padding;
    std::vector<float> weightValues;
    const nvinfer1::PluginField* fields = fc->fields;

    std::cout << "Size of char: " << sizeof(float) << " byte" << std::endl;
    std::cout << "FieldType:kFlOAT32 - " << int(nvinfer1::PluginFieldType::kFLOAT32) << std::endl;
    std::cout << "FieldType:kINT32 - " << int(nvinfer1::PluginFieldType::kINT32) << std::endl;
    for (int i=0; i<fc->nbFields; i++) {
        const char* attrName = fields[i].name;
        std::cout << "FieldName:" << attrName << std::endl;
        std::cout << "FieldType:" << int(fields[i].type) << std::endl;
        std::cout << "FieldLength:" << int(fields[i].length) << std::endl;

        if(strcmp(attrName, "weights") == 0) {
            ASSERT(fields[i].type == nvinfer1::PluginFieldType::kFLOAT32);
            const auto* w = static_cast<const float*>(fields[i].data);
            for (int j = 0; j < weightValues.size(); j++)
            {
                weightValues.push_back(*w);
                w++;
            }

            for (int j = 0; j < 20; j++)
            {
                std::cout << weightValues[j] << ",";
            }
            std::cout << std::endl;

        }
        if(strcmp(attrName, "nbWeights") == 0) {
            ASSERT(fields[i].type == nvinfer1::PluginFieldType::kINT32);
            nbWeights = *(static_cast<const int*>(fields[i].data));
            std::cout  << "nbWeights:" << nbWeights << std::endl;
        }
        if(strcmp(attrName, "filterSize") == 0) {
            ASSERT(fields[i].type == nvinfer1::PluginFieldType::kINT32);
            filterSize = *(static_cast<const int*>(fields[i].data));
            std::cout  << "filterSize:" << filterSize << std::endl;
        }
        if(strcmp(attrName, "nbFilters") == 0) {
            ASSERT(fields[i].type == nvinfer1::PluginFieldType::kINT32);
            nbFilters = *(static_cast<const int*>(fields[i].data));
            std::cout  << "nbFilters:" << nbFilters << std::endl;
        }
        if(strcmp(attrName, "stride") == 0) {
            ASSERT(fields[i].type == nvinfer1::PluginFieldType::kINT32);
            stride = *(static_cast<const int*>(fields[i].data));
            std::cout  << "stride:" << stride << std::endl;
        }
        if(strcmp(attrName, "padding") == 0) {
            ASSERT(fields[i].type == nvinfer1::PluginFieldType::kINT32);
            padding = *(static_cast<const int*>(fields[i].data));
            std::cout  << "padding:" << padding << std::endl;
        }
    }
    nvinfer1::Weights weights{nvinfer1::DataType::kFLOAT, weightValues.data(), (int64_t)weightValues.size()};
    return new Adder2dPlugin(&weights, nbWeights, filterSize, nbFilters, stride, padding);
}

// deserialization plugin implementation
nvinfer1::IPluginV2* Adder2dPluginCreator::deserializePlugin(const char *layerName, const void *serialData, size_t serialLength) {
    return new Adder2dPlugin(serialData, serialLength);
}

const char* Adder2dPluginCreator::getPluginNamespace() const {
    return G_PLUGIN_NAMESPACE;
}

REGISTER_TENSORRT_PLUGIN(Adder2dPluginCreator);
